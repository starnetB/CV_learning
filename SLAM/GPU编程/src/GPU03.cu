#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <sys/time.h>
#include <stdio.h>
#include <math.h>

#define Row  1024
#define Col 1024

__global__ void addKernel(int **C,  int **A, int ** B)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x; //行
    int idy = threadIdx.y + blockDim.y * blockIdx.y;
    if (idx < Col && idy < Row) {   //注意行和列没有逻辑关系
        C[idy][idx] = A[idy][idx] + B[idy][idx];
    }
}

int main()
{
    struct timeval start, end;
    gettimeofday( &start, NULL );
    int **A = (int **)malloc(sizeof(int*) * Row);
    int **B = (int **)malloc(sizeof(int*) * Row);
    int **C = (int **)malloc(sizeof(int*) * Row);
    int *dataA = (int *)malloc(sizeof(int) * Row * Col);
    int *dataB = (int *)malloc(sizeof(int) * Row * Col);
    int *dataC = (int *)malloc(sizeof(int) * Row * Col);

    int **d_A;
    int **d_B;
    int **d_C;
    int *d_dataA;
    int *d_dataB;
    int *d_dataC;

    //malloc device_memory
    hipMalloc((void**)&d_A, sizeof(int *) * Row);
    hipMalloc((void**)&d_B, sizeof(int *) * Row);
    hipMalloc((void**)&d_C, sizeof(int *) * Row);
    hipMalloc((void**)&d_dataA, sizeof(int)*Row*Col);
    hipMalloc((void**)&d_dataB, sizeof(int)*Row*Col);
    hipMalloc((void**)&d_dataC, sizeof(int)*Row*Col);

    //set value
    for (int i = 0; i < Row*Col; i++) {
        dataA[i] = 90;
        dataB[i] = 10;
    }

    //将主机指针A指向设备数据位置，目的是让设备二级指针能够指向设备数据一级指针
    //A 和  dataA 都传到了设备上，但是二者还没有建立对应关系
    for (int i = 0; i < Row; i++) {
        A[i] = d_dataA + Col * i;
        B[i] = d_dataB + Col * i;
        C[i] = d_dataC + Col * i;
    }

    hipMemcpy(d_A, A, sizeof(int*) * Row, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, sizeof(int*) * Row, hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, sizeof(int*) * Row, hipMemcpyHostToDevice);
    hipMemcpy(d_dataA, dataA, sizeof(int) * Row * Col, hipMemcpyHostToDevice);
    hipMemcpy(d_dataB, dataB, sizeof(int) * Row * Col, hipMemcpyHostToDevice);
    

    dim3 threadPerBlock(16, 16);
    dim3 blockNumber((Col + threadPerBlock.x - 1)/threadPerBlock.x, (Row +threadPerBlock.y - 1) / threadPerBlock.y );
    addKernel << <blockNumber, threadPerBlock >> > (d_C, d_A, d_B);

    //拷贝计算数据-一级数据指针
    hipMemcpy(dataC, d_dataC, sizeof(int) * Row * Col, hipMemcpyDeviceToHost);

    int max_error = 0;
    for(int i=0;i<Row*Col;i++)
    {
        //printf("%d\n", dataC[i]);
        max_error += abs(100-dataC[i]);
    }

    //释放内存
    free(A);
    free(B);
    free(C);
    free(dataA);
    free(dataB);
    free(dataC);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_dataA);
    hipFree(d_dataB);
    hipFree(d_dataC);
    printf("max_error is %d\n", max_error);
    gettimeofday( &end, NULL );
    int timeuse = 1000000 * ( end.tv_sec - start.tv_sec ) + end.tv_usec - start.tv_usec;
    printf("total time is %d ms\n", timeuse/1000);
    return 0;
}
