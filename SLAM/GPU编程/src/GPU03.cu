#include "hip/hip_runtime.h"

#include <sys/time.h>
#include <stdio.h>
#include <math.h>

#define Row  1024
#define Col 1024

__global__ void addKernel(int **C,  int **A, int ** B)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x; //行
    int idy = threadIdx.y + blockDim.y * blockIdx.y;
    if (idx < Col && idy < Row) {   //注意行和列没有逻辑关系
        C[idy][idx] = A[idy][idx] + B[idy][idx];
    }
}

int main()
{
    struct timeval start, end;
    gettimeofday( &start, NULL );
    int **A = (int **)malloc(sizeof(int*) * Row);
    int **B = (int **)malloc(sizeof(int*) * Row);
    int **C = (int **)malloc(sizeof(int*) * Row);
    int *dataA = (int *)malloc(sizeof(int) * Row * Col);
    int *dataB = (int *)malloc(sizeof(int) * Row * Col);
    int *dataC = (int *)malloc(sizeof(int) * Row * Col);

    int **d_A;
    int **d_B;
    int **d_C;
    int *d_dataA;
    int *d_dataB;
    int *d_dataC;

    //malloc device_memory
}