#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <iostream>
#include <sys/time.h>

using namespace std;  

__global__ void Plus(float A[], float B[], float C[], int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    C[i] = A[i] + B[i];
}

int main()
{
    struct timeval start, end;
    gettimeofday( &start, NULL );
    float*A, *Ad, *B, *Bd, *C, *Cd;
    int n = 1024 * 1024;
    int size = n * sizeof(float);

    // CPU端分配内存
    A = (float*)malloc(size);
    B = (float*)malloc(size);
    C = (float*)malloc(size);

    // 初始化数组
    for(int i=0;i<n;i++)
    {
    A[i] = 90.0;
    B[i] = 10.0;
    }

    // GPU端分配内存
    hipMalloc((void**)&Ad, size);
    hipMalloc((void**)&Bd, size);
    hipMalloc((void**)&Cd, size);

    // CPU的数据拷贝到GPU端
    hipMemcpy(Ad, A, size, hipMemcpyHostToDevice);
    hipMemcpy(Bd, B, size, hipMemcpyHostToDevice);
    hipMemcpy(Bd, B, size, hipMemcpyHostToDevice);

    // 定义kernel执行配置，（1024*1024/512）个block，每个block里面有512个线程
    dim3 dimBlock(512);
    dim3 dimGrid(n/512);

    // 执行kernel
    Plus<<<dimGrid, dimBlock>>>(Ad, Bd, Cd, n);


    // 将在GPU端计算好的结果拷贝回CPU端
    hipMemcpy(C, Cd, size, hipMemcpyDeviceToHost);

    // 校验误差
    float max_error = 0.0;
    for(int i=0;i<n;i++)
    {
        max_error += fabs(100.0 - C[i]);
    }

    cout << "max error is " << max_error << endl;
    // 释放CPU端、GPU端的内存
    free(A);
    free(B);
    free(C);
    hipFree(Ad);
    hipFree(Bd);
    hipFree(Cd);
    gettimeofday( &end, NULL );
    int timeuse = 1000000 * ( end.tv_sec - start.tv_sec ) + end.tv_usec - start.tv_usec;
    cout << "total time is " << timeuse/1000 << "ms" <<endl;
    return 0;
}